
#include <hip/hip_runtime.h>
#include <iostream>

#define N 268435456

double monotime(){
	timespec ts;
	clock_gettime(CLOCK_MONOTONIC, &ts);
	return ts.tv_sec + ts.tv_nsec * 1e-9;
}

/*
   This kernel(gpu function) will be ran in parallel
   by N different cuda threads
 */
__global__ void vec_add_gpu(float* a, float* b, float* c){
	int i = blockDim.x * blockIdx.x + threadIdx.x; // get thread number
	c[i] = a[i] + b[i];
}

void vec_add_cpu(float* a, float* b, float* c, const int n){
	for(auto i = 0; i < n; i++){
		c[i] = a[i] + b[i];
	}
}

int main(int argc, char** argv){
	int size = N * sizeof(float);
	
	float* a = new float[N];
	float* b = new float[N];
	float* c = new float[N];

	double start, end, gpu_time, cpu_time;
	
	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);
	
	for(int i = 0; i < N; i++){
		a[i] = 1.0f;
		b[i] = 2.0f;
	}
	
	const int threads_per_block = 256;
	const int num_of_blocks = std::ceil(float(N) / threads_per_block);

	start = monotime();	
	
	vec_add_gpu<<< num_of_blocks, threads_per_block >>>(a, b, c);
	hipDeviceSynchronize(); // wait for all threads to finish
	
	end = monotime();
	gpu_time = end - start;

	start = monotime();
	
	vec_add_cpu(a, b, c, N);
	
	end = monotime();
	cpu_time = end - start;
	
	for(int i = 0; i < N; i++){
		if(c[i] != 3){
			std::cout << "error at " << i << " output: " << c[i] << std::endl;
			exit(-1);
		}
	}

	std::cout << "Success" << std::endl;	
	std::cout << "Results over adding a " << N << " size vector" << std::endl;
	std::cout << "cpu time " << cpu_time << std::endl;
	std::cout << "gpu time " << gpu_time << std::endl;

	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}

