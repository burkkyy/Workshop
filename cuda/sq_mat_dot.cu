#include "hip/hip_runtime.h"
#include "cudahelper.hpp"

#define N (1 << 10)

__global__ void dot(float* a, float* b, float* c, const int size){
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	
	//printf("m: %d n: %d\n", m, n);
	//printf("blockDim.x: %d blockDim.y: %d blockIdx.x: %d blockIdx.y: %d threadIdx.x: %d threadIdx.y: %d\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
	
	if(m < N && n < N){
		int temp = 0;
		for(int i = 0; i < size; i++){
			temp += a[m * size + i] * b[i * size + n];
		}
		c[m * size + n] = temp;
	}
}

void dot_cpu(float* a, float* b, float* c, const int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int temp = 0;
			for(int k = 0; k < size; k++){
				temp += a[i * size + k] * b[k * size + j];
			}
			c[i * size + j] = temp;
		}
	}
}

void check_dot(float* a, float* b, float* c, const int size){
	float ans;
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			ans = 0;
			for(int k = 0; k < size; k++){
				ans += a[i * size + k] * b[k * size + j];
			}
			assert(ans == c[i * size + j]);
		}
	}
}

int main(int argc, char** argv){
	double start, end, gpu_time, cpu_time, total_start, total_end, total_time;
	
	total_start = monotime();

	int size = N * N * sizeof(float);
	
	float* a = new float[N*N];
	float* b = new float[N*N];
	float* c = new float[N*N];

	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	const int threads = 16;
	const int blocks = (N + threads - 1) / threads;
	
	dim3 BLOCKS(blocks, blocks);
	dim3 THREADS(threads, threads);
	
	start = monotime();
	dot<<< BLOCKS, THREADS >>>(a, b, c, N);
	hipDeviceSynchronize();
	end = monotime();
	gpu_time = end - start;

	check_dot(a, b, c, N);
	P("gpu passed check");

	start = monotime();
	dot_cpu(a, b, c, N);
	end = monotime();
	cpu_time = end - start;

	check_dot(a, b, c, N);
	P("cpu passed check");
	
	total_end = monotime();
	total_time = total_end - total_start;

	P("Success. Results from taking dot prodcut from size " << N << "x" << N << " matrices:");

#define sec
#ifndef sec
	P("gpu time: " << gpu_time << " ns");
	P("cpu time: " << cpu_time << " ns");
	P("total time: " << total_time << " ns");
#else
	P("gpu time: " << gpu_time * 1e-9 << " sec");
	P("cpu time: " << cpu_time * 1e-9 << " sec");
	P("total time: " << total_time * 1e-9 << " sec");
#endif

	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}

